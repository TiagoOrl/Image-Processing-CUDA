#include "hip/hip_runtime.h"

#include "utils.h"
#include "timer.h"
#include <stdio.h>


__global__
void k_blurChannel (unsigned char * input_channel, unsigned char * outputchannel, int numRows, int numCols) {
    
    int tIdx = threadIdx.x + blockIdx.x * blockDim.x;
    int tIdy = threadIdx.y + blockIdx.y * blockDim.y;

    int index = numCols * tIdx + tIdy;

    if ( index >= numRows * numCols ) return;

    unsigned char new_color = (
        input_channel[index] * 0.5 +                    // itself

        input_channel[index - 1] * 0.125 +              // west
        input_channel[index + 1] * 0.125 +              // east

        input_channel[index - numCols] * 0.125 +        // north
        input_channel[index + numCols] * 0.125 +        // south

        input_channel[index - numCols - 1] * 0.0 +    // northwest
        input_channel[index - numCols + 1] * 0.0 +    // northeast

        input_channel[index + numCols - 1] * 0.0 +    // southwest
        input_channel[index + numCols + 1] * 0.0    // southeast

    );

    outputchannel[index] = new_color;
}

__global__
void k_sobel (
            unsigned char * r_in, unsigned char * g_in, unsigned char * b_in,
            unsigned char * r_out, unsigned char * g_out, unsigned char * b_out,
            int height, int width) {
    
    int tIdx = threadIdx.x + blockIdx.x * blockDim.x;
    int tIdy = threadIdx.y + blockIdx.y * blockDim.y;
    int size = height * width;

    if (tIdx >= width || tIdy >= height) return;

    int index = width * tIdy + tIdx;

    if (
        index - 1 < 0 ||
        index + 1 > size ||
        index - width < 0 ||
        index + width > size ||
        index - width - 1 < 0 ||
        index - width + 1 < 0 ||
        index + width - 1 > size ||
        index + width + 1 > size
    ) return;

    int v_Rkernel = (
        r_in[index] * 0.0 +
                
        r_in[index - 1] * 2.0 +     // west
        r_in[index + 1] * -2.0 +     // east
        
        r_in[index - width] * 0.0 +    // north
        r_in[index + width] * 0.0 +    // south
        
        r_in[index - width - 1] * 1.0 +    // northwest
        r_in[index - width + 1] * -1.0 +    // northeast
        r_in[index + width - 1] * 1.0 +    // southwest
        r_in[index + width + 1] * -1.0 ) ;   // southeast

    int h_Rkernel = (
        r_in[index] * 0.0 +
                
        r_in[index - 1] * 0.0 +     // west
        r_in[index + 1] * 0.0 +     // east
        
        r_in[index - width] * 2.0 +    // north
        r_in[index + width] * -2.0 +    // south
        
        r_in[index - width - 1] * 1.0 +    // northwest
        r_in[index - width + 1] * 1.0 +    // northeast
        r_in[index + width - 1] * -1.0 +    // southwest
        r_in[index + width + 1] * -1.0 ) ;   // southeast

    r_out[index] = (unsigned char) abs( (h_Rkernel + v_Rkernel) / 2);

    int v_Gkernel = (
        g_in[index] * 0.0 +
                
        g_in[index - 1] * 2.0 +     // west
        g_in[index + 1] * -2.0 +     // east
        
        g_in[index - width] * 0.0 +    // north
        g_in[index + width] * 0.0 +    // south
        
        g_in[index - width - 1] * 1.0 +    // northwest
        g_in[index - width + 1] * -1.0 +    // northeast
        g_in[index + width - 1] * 1.0 +    // southwest
        g_in[index + width + 1] * -1.0 ) ;   // southeast

    int h_Gkernel = (
        g_in[index] * 0.0 +
                
        g_in[index - 1] * 0.0 +     // west
        g_in[index + 1] * 0.0 +     // east
        
        g_in[index - width] * 2.0 +    // north
        g_in[index + width] * -2.0 +    // south
        
        g_in[index - width - 1] * 1.0 +    // northwest
        g_in[index - width + 1] * 1.0 +    // northeast
        g_in[index + width - 1] * -1.0 +    // southwest
        g_in[index + width + 1] * -1.0 ) ;   // southeast

    g_out[index] = (unsigned char) abs( (h_Gkernel + v_Gkernel) / 2);

    int v_Bkernel = (
        b_in[index] * 0.0 +
                
        b_in[index - 1] * 2.0 +     // west
        b_in[index + 1] * -2.0 +     // east
        
        b_in[index - width] * 0.0 +    // north
        b_in[index + width] * 0.0 +    // south
        
        b_in[index - width - 1] * 1.0 +    // northwest
        b_in[index - width + 1] * -1.0 +    // northeast
        b_in[index + width - 1] * 1.0 +    // southwest
        b_in[index + width + 1] * -1.0 ) ;   // southeast

    int h_Bkernel = (
        b_in[index] * 0.0 +
                
        b_in[index - 1] * 0.0 +     // west
        b_in[index + 1] * 0.0 +     // east
        
        b_in[index - width] * 2.0 +    // north
        b_in[index + width] * -2.0 +    // south
        
        b_in[index - width - 1] * 1.0 +    // northwest
        b_in[index - width + 1] * 1.0 +    // northeast
        b_in[index + width - 1] * -1.0 +    // southwest
        b_in[index + width + 1] * -1.0 ) ;   // southeast

    b_out[index] = (unsigned char) abs( (h_Bkernel + v_Bkernel) / 2);

}

__global__
void k_sobelBW (
            unsigned char * in_channel, 
            unsigned char * out_channel,
            int height, int width) {
    
    int tIdx = threadIdx.x + blockIdx.x * blockDim.x;
    int tIdy = threadIdx.y + blockIdx.y * blockDim.y;
    int size = width * height;

    if (tIdx >= width || tIdy >= height) return;

    int index = width * tIdy + tIdx;

        if (
        index - 1 < 0 ||
        index + 1 > size ||
        index - width < 0 ||
        index + width > size ||
        index - width - 1 < 0 ||
        index - width + 1 < 0 ||
        index + width - 1 > size ||
        index + width + 1 > size
    ) return;

    int v_Rkernel = (
        in_channel[index] * 0.0 +
                
        in_channel[index - 1] * 2.0 +     // west
        in_channel[index + 1] * -2.0 +     // east
        
        in_channel[index - width] * 0.0 +    // north
        in_channel[index + width] * 0.0 +    // south
        
        in_channel[index - width - 1] * 1.0 +    // northwest
        in_channel[index - width + 1] * -1.0 +    // northeast
        in_channel[index + width - 1] * 1.0 +    // southwest
        in_channel[index + width + 1] * -1.0 ) ;   // southeast

    int h_Rkernel = (
        in_channel[index] * 0.0 +
                
        in_channel[index - 1] * 0.0 +     // west
        in_channel[index + 1] * 0.0 +     // east
        
        in_channel[index - width] * 2.0 +    // north
        in_channel[index + width] * -2.0 +    // south
        
        in_channel[index - width - 1] * 1.0 +    // northwest
        in_channel[index - width + 1] * 1.0 +    // northeast
        in_channel[index + width - 1] * -1.0 +    // southwest
        in_channel[index + width + 1] * -1.0 ) ;   // southeast

    out_channel[index] = (unsigned char) abs( (h_Rkernel + v_Rkernel) / 2);
}


void cuda_sobel( 
    unsigned char * d_inR, unsigned char * d_inG, unsigned char * d_inB,
    unsigned char * d_outR, unsigned char * d_outG, unsigned char * d_outB,
    unsigned char * h_channelR_out, unsigned char * h_channelG_out, unsigned char * h_channelB_out,
    int height, int width, 
    int blockWidth
)   {

    int img_size = width * height;
    int numBlocksX = width / blockWidth + 1;    
    int numBlocksY = height / blockWidth + 1;
    const dim3 threadsPerBlock (blockWidth, blockWidth, 1);
    const dim3 numBlocks (numBlocksX, numBlocksY, 1);
    GpuTimer timer;

    timer.Start();
    k_sobel <<< numBlocks, threadsPerBlock >>> (
        d_inR, d_inG, d_inB,
        d_outR, d_outG, d_outB,
        height, width);

    timer.Stop();
    printf("elapsed: %f ms\n", timer.Elapsed());
    
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());

    checkCudaErrors( hipMemcpy( h_channelR_out, d_outR, sizeof(unsigned char) * img_size, hipMemcpyDeviceToHost) );
    checkCudaErrors( hipMemcpy( h_channelG_out, d_outG, sizeof(unsigned char) * img_size, hipMemcpyDeviceToHost) );
    checkCudaErrors( hipMemcpy( h_channelB_out, d_outB, sizeof(unsigned char) * img_size, hipMemcpyDeviceToHost) );


    hipFree(d_inR);
    hipFree(d_inG);
    hipFree(d_inB);
    
    hipFree(d_outR);
    hipFree(d_outG);
    hipFree(d_outB);
}

void cuda_sobelBW( 
    unsigned char * dIn, 
    unsigned char * dOut, 
    int height, int width, int blockwidth, 
    unsigned char * h_channelOut
)   {

    int numBlocksX = width / blockwidth + 1;    
    int numBlocksY = height / blockwidth + 1;
    const dim3 numBlocks (numBlocksX, numBlocksY, 1);
    const dim3 threadsPerBlock (blockwidth, blockwidth, 1);
    GpuTimer timer;
    

    std::cout << "num of blocks, x = " << numBlocks.x << " y = " << numBlocks.y << std::endl;

    timer.Start();
    k_sobelBW <<< numBlocks, threadsPerBlock >>> (
        dIn, 
        dOut,
        height, width
    );

    timer.Stop();
    printf("elapsed: %f ms\n", timer.Elapsed());
    
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());
    checkCudaErrors( hipMemcpy( h_channelOut, dOut, sizeof(unsigned char) * width * height, hipMemcpyDeviceToHost) );

    hipFree(dIn);
    hipFree(dOut);
}
